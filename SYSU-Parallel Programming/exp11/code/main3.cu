#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#define HEIGHT 4096
#define WIDTH 4096
#define FILTER 3
#define STRIDE 3
#define PAD 1
#define CHANNELS 3
#define OUT_HEIGHT (HEIGHT + 2 * PAD - FILTER) / STRIDE + 1
#define OUT_WIDTH (WIDTH + 2 * PAD - FILTER) / STRIDE + 1

#define checkCUDNN(expression)                                     \
    {                                                              \
        hipdnnStatus_t status = (expression);                       \
        if (status != HIPDNN_STATUS_SUCCESS)                        \
        {                                                          \
            std::cerr << "Error on line " << __LINE__ << ": "      \
                      << hipdnnGetErrorString(status) << std::endl; \
            std::exit(EXIT_FAILURE);                               \
        }                                                          \
    }

int main()
{
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // Initialize input image
    float *image = (float *)malloc(HEIGHT * WIDTH * CHANNELS * sizeof(float));
    for (int i = 0; i < HEIGHT * WIDTH * CHANNELS; ++i)
        image[i] = (float)(rand() % 50) / 100;

    // Tensor descriptors
    hipdnnTensorDescriptor_t input_descriptor, output_descriptor;
    hipdnnFilterDescriptor_t kernel_descriptor;
    hipdnnConvolutionDescriptor_t convolution_descriptor;

    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, CHANNELS, HEIGHT, WIDTH));

    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 1, OUT_HEIGHT, OUT_WIDTH));

    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 1, CHANNELS, FILTER, FILTER));

    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor, PAD, PAD, STRIDE, STRIDE, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn, input_descriptor, kernel_descriptor, convolution_descriptor, output_descriptor, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &convolution_algorithm));

    size_t workspace_bytes = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, input_descriptor, kernel_descriptor, convolution_descriptor, output_descriptor, convolution_algorithm, &workspace_bytes));

    // Memory allocation
    void *d_workspace{nullptr};
    hipMalloc(&d_workspace, workspace_bytes);

    float *d_input{nullptr}, *d_output{nullptr}, *d_kernel{nullptr};
    size_t image_bytes = HEIGHT * WIDTH * CHANNELS * sizeof(float);

    hipMalloc(&d_input, image_bytes);
    hipMemcpy(d_input, image, image_bytes, hipMemcpyHostToDevice);

    hipMalloc(&d_output, image_bytes);
    hipMemset(d_output, 0, image_bytes);

    float h_kernel[FILTER][CHANNELS][FILTER][FILTER];
    for (int kernel = 0; kernel < FILTER; ++kernel)
        for (int channel = 0; channel < CHANNELS; ++channel)
            for (int row = 0; row < FILTER; ++row)
                for (int col = 0; col < FILTER; ++col)
                    h_kernel[kernel][channel][row][col] = (float)(rand() % 50) / 100;

    hipMalloc(&d_kernel, sizeof(h_kernel));
    hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

    const float alpha = 1, beta = 1;

    // Convolution forward
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    checkCUDNN(hipdnnConvolutionForward(cudnn, &alpha, input_descriptor, d_input, kernel_descriptor, d_kernel, convolution_descriptor, convolution_algorithm, d_workspace, workspace_bytes, &beta, output_descriptor, d_output));
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Running time : %f\n", milliseconds / (3.0f * 1000.0f));

    float *h_output = (float *)malloc(image_bytes);
    hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);

    // Output results to file
    FILE *fp = fopen("result.txt", "w");
    for (int i = 0; i < OUT_WIDTH * OUT_HEIGHT; ++i)
        fprintf(fp, "%f ", h_output[i]);

    // Cleanup
    hipFree(d_kernel);
    hipFree(d_input);
    hipFree(d_output);
    free(image);
    free(h_output);
    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroy(cudnn);

    return 0;
}
