#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define mat_height 4096
#define mat_width 4096
#define filter_height 3
#define filter_width 3
#define stride 3

#define block_size_x 32
#define block_size_y 32

#define GET_TIME(now)                           \
    {                                           \
        struct timeval t;                       \
        gettimeofday(&t, NULL);                 \
        now = t.tv_sec + t.tv_usec / 1000000.0; \
    }

int check(float *c, float *d, int n);

#define padding_height ((filter_height / 2) * 2)
#define padding_width ((filter_width / 2) * 2)
#define input_height (mat_height + padding_height)
#define input_width (mat_width + padding_width)

void cpu_convolution(float *output, float *input, float *filter) {
    for (int y = 0; y < mat_height; y += stride) {
        for (int x = 0; x < mat_width; x += stride) {
            for (int i = 0; i < filter_height; i++) {
                for (int j = 0; j < filter_width; j++) {
                    output[y / stride * mat_width + x / stride] += input[(y + i) * input_width + x + j] * filter[i * filter_width + j];
                }
            }
        }
    }
}

void cpu_add(float *arr1, float *arr2, float *arr3, float *result) {
    for (int y = 0; y < mat_height; y += stride) {
        for (int x = 0; x < mat_width; x += stride) {
            int temp = y / stride * mat_width + x / stride;
            result[temp] = arr1[temp] + arr2[temp] + arr3[temp];
        }
    }
}

__global__ void cuda_convolution(float *output, float *input, float *filter) {
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if (y % stride == 0 && x % stride == 0) {
        for (int i = 0; i < filter_height; i++) {
            for (int j = 0; j < filter_width; j++) {
                sum += input[(y + i) * input_width + x + j] * filter[i * filter_width + j];
            }
        }
        output[y / stride * mat_width + x / stride] = sum;
    }
}

__global__ void cuda_add(float *arr1, float *arr2, float *arr3, float *result) {
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (y % stride == 0 && x % stride == 0) {
        result[y / stride * mat_width + x / stride] = arr1[y / stride * mat_width + x / stride] + arr2[y / stride * mat_width + x / stride] + arr3[y / stride * mat_width + x / stride];
    }
}

void init_matrix(float *input, float *filter) {
    for (int i = 0; i < input_height * input_width; i++) {
        input[i] = (float)(rand() % 50) / 100;
    }
    for (int i = 0; i < filter_height * filter_width; i++) {
        filter[i] = (float)(rand() % 50) / 100;
    }
}

int main() {
    double start, finish, time1, time2;

    float *inputs[3], *outputs_cpu[3], *outputs_cuda[3], *filters[3], *result_cpu, *result_cuda;
    for (int i = 0; i < 3; i++) {
        inputs[i] = (float *)malloc(input_height * input_width * sizeof(float));
        outputs_cpu[i] = (float *)malloc(mat_height * mat_width * sizeof(float));
        outputs_cuda[i] = (float *)malloc(mat_height * mat_width * sizeof(float));
        filters[i] = (float *)malloc(filter_height * filter_width * sizeof(float));
        init_matrix(inputs[i], filters[i]);
        memset(outputs_cpu[i], 0, mat_height * mat_width * sizeof(float));
        memset(outputs_cuda[i], 0, mat_height * mat_width * sizeof(float));
    }
    result_cpu = (float *)malloc(mat_height * mat_width * sizeof(float));
    result_cuda = (float *)malloc(mat_height * mat_width * sizeof(float));

    GET_TIME(start);
    for (int i = 0; i < 3; i++) {
        cpu_convolution(outputs_cpu[i], inputs[i], filters[i]);
    }
    cpu_add(outputs_cpu[0], outputs_cpu[1], outputs_cpu[2], result_cpu);
    GET_TIME(finish);
    time1 = finish - start;
    printf("Sequential convolution time:  %f s\n", time1);

    float *d_inputs[3], *d_outputs[3], *d_filters[3], *d_result;
    for (int i = 0; i < 3; i++) {
        hipMalloc((void **)&d_inputs[i], input_height * input_width * sizeof(float));
        hipMalloc((void **)&d_outputs[i], mat_height * mat_width * sizeof(float));
        hipMalloc((void **)&d_filters[i], filter_height * filter_width * sizeof(float));
        hipMemcpy(d_inputs[i], inputs[i], input_height * input_width * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_filters[i], filters[i], filter_height * filter_width * sizeof(float), hipMemcpyHostToDevice);
        hipMemset(d_outputs[i], 0, mat_height * mat_width * sizeof(float));
    }
    hipMalloc((void **)&d_result, mat_height * mat_width * sizeof(float));

    dim3 threads(block_size_x, block_size_y);
    dim3 grid((mat_width + threads.x - 1) / threads.x, (mat_height + threads.y - 1) / threads.y);

    hipDeviceSynchronize();
    GET_TIME(start);
    for (int i = 0; i < 3; i++) {
        cuda_convolution<<<grid, threads>>>(d_outputs[i], d_inputs[i], d_filters[i]);
    }
    cuda_add<<<grid, threads>>>(d_outputs[0], d_outputs[1], d_outputs[2], d_result);
    hipDeviceSynchronize();
    GET_TIME(finish);
    time2 = finish - start;
    printf("CUDA convolution time:        %f s\n", time2);

    hipMemcpy(result_cuda, d_result, mat_height * mat_width * sizeof(float), hipMemcpyDeviceToHost);

    FILE *fp = fopen("result.txt", "w");
    for (int y = 0; y < mat_height; y += stride) {
        for (int x = 0; x < mat_width; x += stride) {
            fprintf(fp, "%f ", result_cuda[y / stride * mat_width + x / stride]);
        }
        fprintf(fp, "\n");
    }
    fclose(fp);

    printf("Running time:      %f s\n",time1+time2);

    if (check(result_cpu, result_cuda, mat_height * mat_width) > 0) {
        printf("The result is wrong.\n");
    } else {
        printf("The result is right.\n");
    }

    for (int i = 0; i < 3; i++) {
        hipFree(d_inputs[i]);
        hipFree(d_outputs[i]);
        hipFree(d_filters[i]);
        free(inputs[i]);
        free(outputs_cpu[i]);
        free(outputs_cuda[i]);
        free(filters[i]);
    }
    hipFree(d_result);
    free(result_cpu);
    free(result_cuda);

    return 0;
}

int check(float *arr1, float *arr2, int n) {
    int errors = 0;
    for (int y = 0; y < mat_height; y += stride) {
        for (int x = 0; x < mat_width; x += stride) {
            int i = y / stride * mat_width + x / stride;
            if (isnan(arr1[i]) || isnan(arr2[i]))
                errors++;
            float diff = (arr1[i] - arr2[i]) / arr1[i];
            if (diff > 1e-6f)
                errors++;
        }
    }
    return errors;
}
