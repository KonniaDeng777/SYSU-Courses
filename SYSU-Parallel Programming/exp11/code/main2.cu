
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define HEIGHT 256
#define WIDTH 256
#define FILTER_SIZE 3
#define STRIDE 1
#define PAD 1
#define CHANNELS 3

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 32

#define GET_TIME(now)                           \
    {                                           \
        struct timeval t;                       \
        gettimeofday(&t, NULL);                 \
        now = t.tv_sec + t.tv_usec / 1000000.0; \
    }

// 并行矩阵乘法函数
__global__ void matMultCUDA(const float *a, const float *b, float *c, int M, int N, int K, int BLOCK_SIZE) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int idx = bid * BLOCK_SIZE + tid;
    int row = idx / M;
    int column = idx % M;
    while (column < K) {
        if (row < M && column < K) {
            float t = 0;
            for (int i = 0; i < N; i++) {
                t += a[row * N + i] * b[i * K + column];
            }
            c[row * K + column] = t;
        }
        column += M;
    }
}

float im2col_get_data(float *im, int row, int col, int channel) { 
    row -= PAD;
    col -= PAD;
    if (row < 0 || col < 0 || row >= HEIGHT || col >= WIDTH) return 0;
    return im[col + WIDTH * (row + HEIGHT * channel)];
}
    
void im2col(float *data_im, float *data_col) {   
    int height_col = (HEIGHT + 2 * PAD - FILTER_SIZE) / STRIDE + 1;    
    int width_col = (WIDTH + 2 * PAD - FILTER_SIZE) / STRIDE + 1;
    int channels_col = CHANNELS * FILTER_SIZE * FILTER_SIZE;    
    for (int c = 0; c < channels_col; ++c) {
        int w_offset = c % FILTER_SIZE;        
        int h_offset = (c / FILTER_SIZE) % FILTER_SIZE;        
        int c_im = c / FILTER_SIZE / FILTER_SIZE;        
        for (int h = 0; h < height_col; ++h) {
            for (int w = 0; w < width_col; ++w) { 
                int im_row = h_offset + h * STRIDE;                
                int im_col = w_offset + w * STRIDE;                
                int col_index = (c * height_col + h) * width_col + w;
                data_col[col_index] = im2col_get_data(data_im, im_row, im_col, c_im);
            }
        }
    }
}

int main() {
    double start, finish, time;
    int height_col = (HEIGHT + 2 * PAD - FILTER_SIZE) / STRIDE + 1;    
    int width_col = (WIDTH + 2 * PAD - FILTER_SIZE) / STRIDE + 1;    
    int channels_col = CHANNELS * FILTER_SIZE * FILTER_SIZE;    
    
    float *im = (float *)malloc(HEIGHT * WIDTH * CHANNELS * sizeof(float));
    float *col = (float *)malloc(channels_col * height_col * width_col * sizeof(float));
    float *filter = (float *)malloc(CHANNELS * FILTER_SIZE * FILTER_SIZE * sizeof(float));
    float *c = (float *)malloc(CHANNELS * width_col * height_col * sizeof(float));

    for (int i = 0; i < HEIGHT * WIDTH * CHANNELS; i++) {
        im[i] = (float)(rand() % 50) / 100;
    }
    for (int i = 0; i < FILTER_SIZE * FILTER_SIZE * CHANNELS; i++) {
        filter[i] = (float)(rand() % 50) / 100;
    }

    GET_TIME(start);

    im2col(im, col);

    float *cuda_a, *cuda_b, *cuda_c;
    hipMalloc((void **)&cuda_a, sizeof(float) * CHANNELS * FILTER_SIZE * FILTER_SIZE);
    hipMalloc((void **)&cuda_b, sizeof(float) * channels_col * height_col * width_col);
    hipMalloc((void **)&cuda_c, sizeof(float) * CHANNELS * width_col * height_col);

    hipMemcpy(cuda_a, filter, sizeof(float) * CHANNELS * FILTER_SIZE * FILTER_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, col, sizeof(float) * channels_col * height_col * width_col, hipMemcpyHostToDevice);

    int BLOCK_SIZE = HEIGHT;
    int blocks_num = (CHANNELS * width_col * height_col + BLOCK_SIZE - 1) / BLOCK_SIZE;

    matMultCUDA<<<blocks_num, BLOCK_SIZE>>>(cuda_a, cuda_b, cuda_c, CHANNELS, FILTER_SIZE * FILTER_SIZE, width_col * height_col, BLOCK_SIZE);

    hipMemcpy(c, cuda_c, sizeof(float) * CHANNELS * width_col * height_col, hipMemcpyDeviceToHost);

    GET_TIME(finish);
    time = finish - start;

    FILE *fp = fopen("result.txt", "w");
    for (int i = 0; i < CHANNELS * width_col * height_col; i++) {
        fprintf(fp, "%f ", c[i]);
    }
    fclose(fp);

    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);
    free(c);
    free(im);
    free(col);

    printf("Running time: %f s\n\n", time);

    return 0;
}
