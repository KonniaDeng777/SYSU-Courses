#include <hip/hip_runtime.h>

#include <iostream>
#include <chrono>
#include <cstdlib>

// CUDA核函数用于矩阵乘法
__global__ void matrixMulKernel(float* A, float* B, float* C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        float value = 0;
        for (int e = 0; e < n; ++e) {
            value += A[row * n + e] * B[e * k + col];
        }
        C[row * k + col] = value;
    }
}

// 矩阵初始化函数
void initializeMatrix(float* matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

// 矩阵乘法主函数
void matrixMulCUDA(int m, int n, int k, int blockSize) {
    size_t sizeA = m * n * sizeof(float);
    size_t sizeB = n * k * sizeof(float);
    size_t sizeC = m * k * sizeof(float);

    // 分配主机内存
    float* h_A = (float*)malloc(sizeA);
    float* h_B = (float*)malloc(sizeB);
    float* h_C = (float*)malloc(sizeC);

    // 初始化矩阵A和B
    initializeMatrix(h_A, m, n);
    initializeMatrix(h_B, n, k);

    // 分配设备内存
    float* d_A; hipMalloc(&d_A, sizeA);
    float* d_B; hipMalloc(&d_B, sizeB);
    float* d_C; hipMalloc(&d_C, sizeC);

    // 将主机内存数据拷贝到设备内存
    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    // 定义CUDA网格和块结构
    dim3 blockDim(blockSize, blockSize);
    dim3 gridDim((k + blockSize - 1) / blockSize, (m + blockSize - 1) / blockSize);

    // 记录开始时间
    auto start = std::chrono::high_resolution_clock::now();

    // 调用矩阵乘法核函数
    matrixMulKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, m, n, k);

    // 同步设备
    hipDeviceSynchronize();

    // 记录结束时间
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration = end - start;

    // 将结果从设备内存拷贝回主机内存
    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    // 输出计算时间
    std::cout << "Running time: " << duration.count() << " ms" << std::endl;

    // 释放内存
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int m, n, k, blockSize;

    // 输入矩阵的维度和线程块大小
    std::cout << "Enter m,n,k: ";
    std::cin >> m;
    std::cin >> n;
    std::cin >> k;
    std::cout << "Enter the block size: ";
    std::cin >> blockSize;

    // 运行矩阵乘法
    matrixMulCUDA(m, n, k, blockSize);

    return 0;
}
