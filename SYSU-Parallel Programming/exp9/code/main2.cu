#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

__global__ void transpose(int *A, int *AT, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        AT[row * n + col] = A[col * n + row];
    }
}

int main() {
    int n;
    std::cout << "Enter the size of the square matrix (between 512 and 2048): ";
    std::cin >> n;

    if (n < 512 || n > 2048) {
        std::cerr << "Size must be between 512 and 2048!" << std::endl;
        return 1;
    }

    int *h_A = new int[n * n];
    int *h_AT = new int[n * n];

    srand(time(NULL));
    for (int i = 0; i < n * n; ++i) {
        h_A[i] = rand() % 100; 
    }

    int *d_A, *d_AT;
    hipMalloc((void**)&d_A, n * n * sizeof(int));
    hipMalloc((void**)&d_AT, n * n * sizeof(int));

    hipMemcpy(d_A, h_A, n * n * sizeof(int), hipMemcpyHostToDevice);

    int block_size;
    std::cout << "Enter the block size for CUDA kernel: ";
    std::cin >> block_size;
    dim3 blockSize(block_size, block_size);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y);

    clock_t start = clock();
    transpose<<<gridSize, blockSize>>>(d_A, d_AT, n);
    hipDeviceSynchronize();
    clock_t end = clock();
    double time_taken = double(end - start) / CLOCKS_PER_SEC;

    hipMemcpy(h_AT, d_AT, n * n * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Matrix A:" << std::endl;
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cout << h_A[i * n + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "Transpose of A :" << std::endl;
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cout << h_AT[i * n + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "Running time : " << time_taken << " seconds" << std::endl;

    delete[] h_A;
    delete[] h_AT;
    hipFree(d_A);
    hipFree(d_AT);

    return 0;
}
