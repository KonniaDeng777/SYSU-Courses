#include <iostream>
#include <hip/hip_runtime.h>

__global__ void helloFromGPU(int m, int k) {
    int blockId = blockIdx.x;
    int threadIdX = threadIdx.x;
    int threadIdY = threadIdx.y;
    printf("Hello World from Thread (%d, %d) in Block %d!\n", threadIdX, threadIdY, blockId);
}

int main() {
    int n, m, k;
    std::cout << "Enter the values for n (number of blocks), m (threads in x dimension), k (threads in y dimension): ";
    std::cin >> n >> m >> k;
    dim3 threadsPerBlock(m, k);
    helloFromGPU<<<n, threadsPerBlock>>>(m, k);
    hipDeviceSynchronize();
    std::cout << "Hello World from the host!" << std::endl;
    return 0;
}
