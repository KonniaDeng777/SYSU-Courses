/*
nvcc main4.cu -o main4
./main4
*/
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void multiplyMatricesKernel(int *A, int *B, int *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

void multiplyMatricesCUDA(const vector<vector<int>> &A, const vector<vector<int>> &B, vector<vector<int>> &C) {
    int n = A.size();
    int size = n * n * sizeof(int);

    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, &A[0][0], size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, &B[0][0], size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((n + 15) / 16, (n + 15) / 16);

    multiplyMatricesKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);

    hipMemcpy(&C[0][0], d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int n = 500;
    vector<vector<int>> A(n, vector<int>(n, 1));
    vector<vector<int>> B(n, vector<int>(n, 1));
    vector<vector<int>> C(n, vector<int>(n, 0));

    multiplyMatricesCUDA(A, B, C);

    return 0;
}
